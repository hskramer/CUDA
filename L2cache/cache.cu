#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <stdlib.h>

#define MEGABYTE 1048576

__global__ void kernel(const int *in, int *out)
{
	out[0 + threadIdx.x] = in[0 + threadIdx.x];
}

int main(int argc, char* argv[])
{

	int *dev_in = 0;
	int *dev_out = 0;

	int * in = (int*)malloc(sizeof(int) * MEGABYTE);
	int * out = (int*)malloc(sizeof(int) * MEGABYTE);

	for (int i = 0; i < MEGABYTE; i++)
		in[i] = i;

	hipMalloc((void**)&dev_in, sizeof(int) * MEGABYTE);
	hipMalloc((void**)&dev_out, sizeof(int) * MEGABYTE);

	hipProfilerStart();

	hipMemcpy(dev_in, in, sizeof(int) * MEGABYTE, hipMemcpyHostToDevice);

	kernel << <1, 32 >> >(dev_in, dev_out);

	hipMemcpy(out, dev_out, sizeof(int) * MEGABYTE, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipProfilerStop();


	free(in);
	free(out);
	hipFree(dev_in);
	hipFree(dev_out);
	hipDeviceReset();

	return 0;
}