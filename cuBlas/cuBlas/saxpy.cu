#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__
void init(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += stride)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

}

int main(void)
{
	hipblasStatus_t		stat;
	hipblasHandle_t		handle;

	hipDeviceProp_t		prop;
	hipEvent_t			start, stop;

	int		N = 1 << 20;
	int		numSM;

	float	*d_x, *d_y, *x, *y;
	float	 elapsed;

	const float a = 2.0f;


	checkCuda(hipEventCreate(&start));
	checkCuda(hipEventCreate(&stop));
	checkCuda(hipEventRecord(start, 0));

	checkCuda(hipMalloc(&d_x, N * sizeof(float)));
	checkCuda(hipMalloc(&d_y, N * sizeof(float)));

	checkCuda(hipHostAlloc(&x, N * sizeof(float), hipHostMallocDefault));
	checkCuda(hipHostAlloc(&y, N * sizeof(float), hipHostMallocDefault));

	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);


	checkCuda(hipGetDeviceProperties(&prop, 0));
	numSM = prop.multiProcessorCount;

	init << <64 * numSM, 1024 >> > (N, x, y);

	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("CUBLAS initialization failed\n");
		return	EXIT_FAILURE;
	}

	hipblasSetVector(N, sizeof(x[0]), x, 1, d_x, 1);
	hipblasSetVector(N, sizeof(y[0]), y, 1, d_y, 1);

	stat = hipblasSaxpy(handle, N, &a, d_x, 1, d_y, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("hipblasSaxpy failed\n");
		return EXIT_FAILURE;
	}


	// Wait for gpu to finsih before allowing host to access the memory

	hipDeviceSynchronize();

	hipblasGetVector(N, sizeof(y[0]), d_y, 1, y, 1);


	checkCuda(hipEventRecord(stop, 0));
	checkCuda(hipEventSynchronize(stop));
	checkCuda(hipEventElapsedTime(&elapsed, start, stop));

	float maxError = 0.0f;

	for (int i = 0; i < N; i++)
		maxError = max(maxError, abs(y[i] - 4.0f));

	printf("Max error: %f\n", maxError);
	printf("Time to calculate: %3.2fms, this does not include the time calculating the error\n", elapsed);

	hipFree(x);
	hipFree(y);

	return 0;

}