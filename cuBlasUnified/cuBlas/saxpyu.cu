#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


__global__
void init(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += stride)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

}


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}


int main(void)
{
	hipblasStatus_t		stat;
	hipblasHandle_t		handle;

	hipDeviceProp_t		prop;
	hipEvent_t			start, stop;

	int		N = 1 << 20;	
	int		numSM;

	float	*d_x, *d_y, *x, *y;
	float	 elapsed;

	const float a = 2.0f;

	
	checkCuda(hipEventCreate(&start));
	checkCuda(hipEventCreate(&stop));
	checkCuda(hipEventRecord(start, 0));

	checkCuda(hipMallocManaged(&d_x, N*sizeof(float)));
	checkCuda(hipMallocManaged(&d_y, N*sizeof(float)));

	checkCuda(hipMallocManaged(&x, N * sizeof(float)));
	checkCuda(hipMallocManaged(&y, N * sizeof(float)));

	
	checkCuda(hipGetDeviceProperties(&prop, 0));
	numSM = prop.multiProcessorCount;

	init <<<64 * numSM, 1024 >> > (N, x, y);

	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("CUBLAS initialization failed\n");
		return	EXIT_FAILURE;
	}

	hipblasSetVector(N, sizeof(x[0]), x, 1, d_x, 1);
	hipblasSetVector(N, sizeof(y[0]), y, 1, d_y, 1);

	//The new CUBLAS library now passes constants by reference not value
	stat = hipblasSaxpy(handle, N, &a, d_x, 1, d_y, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("hipblasSaxpy failed\n");
		return EXIT_FAILURE;
	}
	

	// Wait for gpu to finsih before allowing host to access the memory

	hipDeviceSynchronize();

	hipblasGetVector(N, sizeof(y[0]), d_y, 1, y, 1);
	

	checkCuda(hipEventRecord(stop, 0));
	checkCuda(hipEventSynchronize(stop));
	checkCuda(hipEventElapsedTime(&elapsed, start, stop));

	float maxError = 0.0f;

	for (int i = 0; i < N; i++)
		maxError = max(maxError, abs(y[i] - 4.0f));

	printf("Max error: %f\n", maxError);
	printf("Time to calculate: %3.2fms, this does not include the time calculating the error\n", elapsed);

	hipFree(x);
	hipFree(y);

	return 0;

}